#include "hip/hip_runtime.h"
#include "LogDemonsRegGPU.cuh"

void LogDemonsRegGPU::printDeviceProp(int deviceID){
	printf("CUDA Device Query...\n");
	printf("CUDA Device #%d\n", deviceID);

	hipDeviceProp_t devProp;
	hipGetDeviceProperties(&devProp, deviceID);

	printf("Major revision number:         %d\n", devProp.major);
	printf("Minor revision number:         %d\n", devProp.minor);
	printf("Name:                          %s\n", devProp.name);
	printf("Total global memory:           %u\n", devProp.totalGlobalMem);
	printf("Total shared memory per block: %u\n", devProp.sharedMemPerBlock);
	printf("Total registers per block:     %d\n", devProp.regsPerBlock);
	printf("Warp size:                     %d\n", devProp.warpSize);
	printf("Maximum memory pitch:          %u\n", devProp.memPitch);
	printf("Maximum threads per block:     %d\n", devProp.maxThreadsPerBlock);
	for (int i = 0; i < 3; ++i)
		printf("Maximum dimension %d of block:  %d\n", i, devProp.maxThreadsDim[i]);
	for (int i = 0; i < 3; ++i)
		printf("Maximum dimension %d of grid:   %d\n", i, devProp.maxGridSize[i]);
	printf("Clock rate:                    %d\n", devProp.clockRate);
	printf("Total constant memory:         %u\n", devProp.totalConstMem);
	printf("Texture alignment:             %u\n", devProp.textureAlignment);
	printf("Concurrent copy and execution: %s\n", (devProp.deviceOverlap ? "Yes" : "No"));
	printf("Number of multiprocessors:     %d\n", devProp.multiProcessorCount);
	printf("Kernel execution timeout:      %s\n", (devProp.kernelExecTimeoutEnabled ? "Yes" : "No"));
	return;
}

void LogDemonsRegGPU::getCudaError(const char* s){
	hipError_t cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess){
		cerr << s << " failed!" << endl;
		cerr << hipGetErrorString(cudaStatus) << endl;
		getchar();
		exit(1);
	}
	else
	{
		if (debug)
			cout << s << " success" << endl;
	}
}

hipTextureObject_t LogDemonsRegGPU::initGaussValue(float* &h_gaussianVal, float* &d_gaussVal, float sigma, int &radius){

	// Compute Gaussian Values
	radius = int(ceil(3.0f *sigma));
	h_gaussianVal = new float[radius + 1];
	float sum = 0;
	for (int i = 0; i < radius + 1; ++i){
		h_gaussianVal[i] = exp(-(i*i) / (2 * sigma));
		i == 0 ? sum += h_gaussianVal[i] / 2.0 : sum += h_gaussianVal[i];
	}
	for (int i = 0; i < radius + 1; ++i){
		h_gaussianVal[i] /= (2 * sum);
	}

	// copy the gaussian values to the device
	hipMalloc((void**)&d_gaussVal, sizeof(float)*(radius + 1));
	hipMemcpy(d_gaussVal, h_gaussianVal, sizeof(float)*(radius + 1), hipMemcpyHostToDevice);
	getCudaError("CreateGaussianMemory");


	// create texture object for fast access
	hipResourceDesc h_resDesc;
	memset(&h_resDesc, 0, sizeof(h_resDesc));
	h_resDesc.resType = hipResourceTypeLinear;
	h_resDesc.res.linear.devPtr = d_gaussVal;
	h_resDesc.res.linear.desc.f = hipChannelFormatKindFloat;
	h_resDesc.res.linear.desc.x = 32;
	h_resDesc.res.linear.sizeInBytes = (radius+1)*sizeof(float);
	

	hipTextureDesc h_texDesc;
	memset(&h_texDesc, 0, sizeof(h_texDesc));
	h_texDesc.readMode = hipReadModeElementType;
	h_texDesc.filterMode = hipFilterModePoint;

	hipTextureObject_t tex;
	hipCreateTextureObject(&tex, &h_resDesc, &h_texDesc, NULL);
	getCudaError("CreateGaussianTextureObject");

	return tex;
}

hipTextureObject_t LogDemonsRegGPU::CreateTextureObject(float* d_I, hipArray *d_cuArr){
	hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();
	hipMemcpy3DParms copyParams = { 0 };

	//Copy array form d_I to  hipArray d_cuArr
	copyParams.srcPtr = make_hipPitchedPtr(
		d_I,
		sizeof(float)*dim[0],
		dim[0],
		dim[1]);

	copyParams.dstArray = d_cuArr;
	copyParams.extent = make_hipExtent(
		dim[0],
		dim[1],
		dim[2]);

	copyParams.kind = hipMemcpyDeviceToDevice;
	hipMemcpy3D(&copyParams);
	hipResourceDesc    texRes;
	memset(&texRes, 0, sizeof(hipResourceDesc));
	texRes.resType = hipResourceTypeArray;
	texRes.res.array.array = d_cuArr;
	hipTextureDesc     texDescr;
	memset(&texDescr, 0, sizeof(hipTextureDesc));
	texDescr.normalizedCoords = false;
	texDescr.filterMode = hipFilterModePoint;
	texDescr.addressMode[0] = hipAddressModeClamp;
	texDescr.addressMode[1] = hipAddressModeClamp;
	texDescr.addressMode[2] = hipAddressModeClamp;
	texDescr.readMode = hipReadModeElementType;
	hipTextureObject_t texObj;
	hipCreateTextureObject(&texObj, &texRes, &texDescr, NULL);
	return texObj;
}


void LogDemonsRegGPU::initialize(){
	// initialize the CPU memory first
	LogDemonsReg::initialize();
	d3_dim = dim3(dim[0], dim[1], dim[2]);

	// Initialize GPU memory for images
	hipMalloc((void**)&d_fixed, sizeof(float)*len);
	hipMemcpy(d_fixed, fixed, sizeof(float)*len, hipMemcpyHostToDevice);

	hipMalloc((void**)&d_moving, sizeof(float)*len);
	hipMemcpy(d_moving, moving, sizeof(float)*len, hipMemcpyHostToDevice);
	hipMalloc((void**)&d_deformedMoving, sizeof(float)*len);
	hipMemcpy(d_deformedMoving, deformedMoving, sizeof(float)*len, hipMemcpyHostToDevice);
	getCudaError("GPU Image initialization");

	// Initialize GPU memory for vector field
	hipMalloc((void**)&d_ux, sizeof(float)*len);
	hipMemset(d_ux, 0, sizeof(float)*len);
	hipMalloc((void**)&d_uy, sizeof(float)*len);
	hipMemset(d_uy, 0, sizeof(float)*len);
	hipMalloc((void**)&d_uz, sizeof(float)*len);
	hipMemset(d_uz, 0, sizeof(float)*len);
	hipMalloc((void**)&d_vx, sizeof(float)*len);
	hipMemset(d_vx, 0, sizeof(float)*len);
	hipMalloc((void**)&d_vy, sizeof(float)*len);
	hipMemset(d_vy, 0, sizeof(float)*len);
	hipMalloc((void**)&d_vz, sizeof(float)*len);
	hipMemset(d_vz, 0, sizeof(float)*len);
	hipMalloc((void**)&d_sx, sizeof(float)*len);
	hipMemset(d_sx, 0, sizeof(float)*len);
	hipMalloc((void**)&d_sy, sizeof(float)*len);
	hipMemset(d_sy, 0, sizeof(float)*len);
	hipMalloc((void**)&d_sz, sizeof(float)*len);
	hipMemset(d_sz, 0, sizeof(float)*len);
	getCudaError("GPU Vector field Memory initialization");

	// Initialize GPU memory for temp memory
	hipMalloc((void**)&d_uxf, sizeof(float)*len);
	hipMemset(d_uxf, 0, sizeof(float)*len);
	hipMalloc((void**)&d_uyf, sizeof(float)*len);
	hipMemset(d_uyf, 0, sizeof(float)*len);
	hipMalloc((void**)&d_uzf, sizeof(float)*len);
	hipMemset(d_uzf, 0, sizeof(float)*len);
	hipMalloc((void**)&d_en, sizeof(float)*len);
	hipMemset(d_en, 0, sizeof(float)*len);

	// Initialize GPU 3D Array memory
	hipChannelFormatDesc channelDescfloat = hipCreateChannelDesc<float>();
	hipMalloc3DArray(&d_cuArr_mov, &channelDescfloat, make_hipExtent(dim[0], dim[1], dim[2]),	0);
	hipMalloc3DArray(&d_cuArr_vx,  &channelDescfloat, make_hipExtent(dim[0], dim[1], dim[2]), 0);
	hipMalloc3DArray(&d_cuArr_vy,  &channelDescfloat, make_hipExtent(dim[0], dim[1], dim[2]), 0);	
	hipMalloc3DArray(&d_cuArr_vz,  &channelDescfloat, make_hipExtent(dim[0], dim[1], dim[2]), 0);

	// Create moving texture for interpolation
	tex_mov = CreateTextureObject(d_moving, d_cuArr_mov);

	// Precompute fixed image gradient
	gradient(d_fixed, d_uxf, d_uyf, d_uzf);

	// Initialize Guassian Values and put into texture memory
	tex_gauss_f = initGaussValue(gaussian_f, d_gaussian_f, opt.sigma_f, radius_f);
	tex_gauss_d = initGaussValue(gaussian_d, d_gaussian_d, opt.sigma_d, radius_d);

	getCudaError("GPU Gaussian Value initialization");
}

void LogDemonsRegGPU::Register(){
	LogDemonsRegGPU::initialize();

	for (int iter = 0; iter < opt.iteration_max; ++iter) {

		/*	Given the current transformation s, compute a correspondence update field u
		by mimimizing E(u) w.r.t. u		*/
		findupdate();
		cout << "findupdate" << endl;

		/*	For a fluid-like regularization let u <- K(sigma_f) * u	*/
		this->imgaussian(d_ux, d_uy, d_uz, tex_gauss_f, radius_f);
		cout << "imgaussian 0 " << endl;

		/*	Let v <- v compose u	*/
		compose();
		cout << "compose " << endl;

		/*	For a diffusion like regularization let s <- K(sigma_d)*c (else, s<-c)	*/
		imgaussian(d_vx, d_vy, d_vz, tex_gauss_d, radius_d);
		cout << "imgaussian 1 " << endl;

		/*	s = exp(v)	*/
		expfield();
		cout << "expfield " << endl;

		//Transform the moving image
		iminterpolate();
		cout << "iminterpolate " << endl;
		//evulate energy
		energy_vec.push_back(energy());
		cout << "energy " << endl;
		printf("Iteration %i - Energy: %f\n", iter + 1, energy_vec.back());

		if (iter > 4){
			if ((energy_vec[iter - 5] - energy_vec[iter]) < (energy_vec[0] * opt.stop_criterium)){
				printf("e-5: %f\n", energy_vec[iter - 5]);
				printf("e: %f\n", energy_vec[iter]);
				printf("e-5 - e: %f\n", energy_vec[iter - 5] - energy_vec[iter]);
				printf("e[0] * opt.stop_criterium: %f\n", energy_vec[0] * opt.stop_criterium);
				//break;
			}
		}
		std::string filename = "C:\\Users\\Martin\\Documents\\gpu_diffeomorphic_logdemons_private\\test_data\\register_results\\gpu_results\\Mp_" + to_string(iter + 1) + std::string(".bin");
		saveImage<float>(deformedMoving, dim, filename.c_str());
	}

	printf("LogDemonsRegGPU Complete\n");	printf("\n"); printf("\n");
	for (int iter = 0; iter < energy_vec.size(); ++iter){
		printf("Iteration %i - Energy: %f\n", iter + 1, energy_vec[iter]);
	}
}

void LogDemonsRegGPU::gradient(float* d_I, float* d_fx, float* d_fy, float* d_fz){
	dim3 blocksize = dim3(8, 8, 8);
	dim3 gridsize = dim3(ceil(double(dim[0]) / double(blocksize.x)), ceil(double(dim[1]) / double(blocksize.y)), ceil(double(dim[2]) / double(blocksize.z)));

	if (debug){
		cout << "Gradient decomposing image..." << endl;
		cout << "blocksize: " << blocksize.x << " " << blocksize.y << " " << blocksize.z << " " << endl;
		cout << "gridsize: " << gridsize.x << " " << gridsize.y << " " << gridsize.z << " " << endl;
	}

	gradientKernel KERNEL_ARGS2(gridsize, blocksize) (d_I, d_fx, d_fy, d_fz, d3_dim);
	getCudaError("gradientKernel");
	if (debug) {
		hipDeviceSynchronize();
		getCudaError("DeviceSynchorize gradientKernel");
	}
}

float LogDemonsRegGPU::thrustFindMaxElement(float* d_f){
	// Use thrust to get the max element
	thrust::device_vector<float> dev_vec(d_f, d_f + len);
	thrust::device_vector<float>::iterator iter = thrust::max_element(dev_vec.begin(), dev_vec.end());
	float max_val = *iter;
	return max_val;
}

void LogDemonsRegGPU::findupdate(){
	gradient(d_deformedMoving, d_ux, d_uy, d_uz);
	float alpha2 = (opt.sigma_i*opt.sigma_i) / (opt.sigma_x*opt.sigma_x);

	dim3 blocksize = dim3(1024, 1, 1);
	dim3 gridsize = dim3(ceil(float(len) / float(blocksize.x)), 1, 1);

	if (debug){
		cout << "Computing update kernel..." << endl;
		cout << "blocksize: " << blocksize.x << " " << blocksize.y << " " << blocksize.z << " " << endl;
		cout << "gridsize: " << gridsize.x << " " << gridsize.y << " " << gridsize.z << " " << endl;
	}
	normalizeVectorKernel KERNEL_ARGS2(gridsize, blocksize) (d_ux, d_uy, d_uz, d_normg2, len);
	getCudaError("normalizeVectorKernel");
	if (debug) {
		hipDeviceSynchronize();
		getCudaError("DeviceSynchorize normalizeVectorKernel");
	}

	updateKernel KERNEL_ARGS2(gridsize, blocksize) (d_fixed, d_deformedMoving, d_ux, d_uy, d_uz, d_uxf, d_uyf, d_uzf, alpha2, d3_dim);

	getCudaError("updateKernel");
	if (debug) {
		hipDeviceSynchronize();
		getCudaError("DeviceSynchorize updateKernel");
	}
}

void LogDemonsRegGPU::imgaussian(float* d_fx, float* d_fy, float* d_fz, hipTextureObject_t tex, int radius){
	
	if (maxdim(marginSize) < radius) printf("Warning: Margin size (%u) is smaller than the gaussian radius (%i). Increase the padding factor. \n", maxdim(marginSize), radius);
		
	//X-direction
	dim3 blocksize;
	dim3 gridsize;
	int shsize;
	
	blocksize=dim3(dim[0], 1, 1);
	gridsize= dim3(dim[1], dim[2],1);
	shsize = sizeof(float4)*(blocksize.x + radius * 2);

	if (debug){
		cout << "Gaussian blur... X direction" << endl;
		cout << "blocksize: " << blocksize.x << " " << blocksize.y << " " << blocksize.z << " " << endl;
		cout << "gridsize: " << gridsize.x << " " << gridsize.y << " " << gridsize.z << " " << endl;
	}

	gaussianKernel KERNEL_ARGS3(gridsize, blocksize, shsize) (d_fx, tex, d3_dim, radius, 0);
	gaussianKernel KERNEL_ARGS3(gridsize, blocksize, shsize) (d_fy, tex, d3_dim, radius, 0);
	gaussianKernel KERNEL_ARGS3(gridsize, blocksize, shsize) (d_fz, tex, d3_dim, radius, 0);
	getCudaError("gaussianKernel - X");
	if (debug){
		hipDeviceSynchronize();
		getCudaError("DeviceSynchorize Gaussian X");
	}
	//Y-direciton
	blocksize = dim3{ 32, 32, 1 };
	gridsize = dim3{ (unsigned int)ceil(dim[0] / 32.0), dim[2], 1 };
	shsize = 32 * dim[1] * sizeof(float);

	if (debug){
		cout << "Gaussian blur... Y direction" << endl;
		cout << "blocksize: " << blocksize.x << " " << blocksize.y << " " << blocksize.z << " " << endl;
		cout << "gridsize: " << gridsize.x << " " << gridsize.y << " " << gridsize.z << " " << endl;
	}

	gaussianKernel KERNEL_ARGS3(gridsize, blocksize, shsize) (d_fx, tex, d3_dim, radius, 1);
	gaussianKernel KERNEL_ARGS3(gridsize, blocksize, shsize) (d_fy, tex, d3_dim, radius, 1);
	gaussianKernel KERNEL_ARGS3(gridsize, blocksize, shsize) (d_fz, tex, d3_dim, radius, 1);
	getCudaError("gaussianKernel - Y");
	if (debug){
		hipDeviceSynchronize();
		getCudaError("DeviceSynchorize Gaussian Y");
	}
	//Z-direciton
	blocksize = dim3{ 32, 32, 1 };
	gridsize = dim3{ (unsigned int)ceil(dim[0] / 32.0), dim[1], 1 };
	shsize = 32 * dim[2] * sizeof(float);

	if (debug){
		cout << "Gaussian blur... Z direction" << endl;
		cout << "blocksize: " << blocksize.x << " " << blocksize.y << " " << blocksize.z << " " << endl;
		cout << "gridsize: " << gridsize.x << " " << gridsize.y << " " << gridsize.z << " " << endl;
	}
	gaussianKernel KERNEL_ARGS3(gridsize, blocksize, shsize) (d_fx, tex, d3_dim, radius, 2);
	gaussianKernel KERNEL_ARGS3(gridsize, blocksize, shsize) (d_fy, tex, d3_dim, radius, 2);
	gaussianKernel KERNEL_ARGS3(gridsize, blocksize, shsize) (d_fz, tex, d3_dim, radius, 2);
	getCudaError("gaussianKernel - Z");
		if (debug){
		hipDeviceSynchronize();
		getCudaError("DeviceSynchorize Gaussian Z");
	}
}

void LogDemonsRegGPU::compose(){
	dim3 blocksize;
	dim3 gridsize;

	blocksize = dim3(dim[0], 1, 1);
	gridsize = dim3(dim[1], dim[2], 1);
	coordinateImageKernel KERNEL_ARGS2(gridsize, blocksize) (d_vx, d_vy, d_vz, d3_dim, addCoordinate);
	coordinateImageKernel KERNEL_ARGS2(gridsize, blocksize) (d_ux, d_uy, d_uz, d3_dim, addCoordinate);

	tex_vx = CreateTextureObject(d_vx, d_cuArr_vx);
	tex_vy = CreateTextureObject(d_vy, d_cuArr_vx);
	tex_vz = CreateTextureObject(d_vz, d_cuArr_vx);

	interpolate(tex_vx, ux, uy, uz, vx);
	interpolate(tex_vy, ux, uy, uz, vy);
	interpolate(tex_vz, ux, uy, uz, vz);

	coordinateImageKernel KERNEL_ARGS2(gridsize, blocksize) (d_vx, d_vy, d_vz, d3_dim, substractCoordinate);


	hipDestroyTextureObject(tex_vx);
	getCudaError("destroyTextureObject");
	hipDestroyTextureObject(tex_vy);
	getCudaError("destroyTextureObject");
	hipDestroyTextureObject(tex_vz);
	getCudaError("destroyTextureObject");
}

void LogDemonsRegGPU::self_compose(){
	dim3 blocksize;
	dim3 gridsize;

	blocksize = dim3(dim[0], 1, 1);
	gridsize = dim3(dim[1], dim[2], 1);
	coordinateImageKernel KERNEL_ARGS2(gridsize, blocksize) (d_sx, d_sy, d_sz, d3_dim, addCoordinate);

	tex_vx = CreateTextureObject(d_sx, d_cuArr_vx);
	tex_vy = CreateTextureObject(d_sy, d_cuArr_vx);
	tex_vz = CreateTextureObject(d_sz, d_cuArr_vx);

	interpolate(tex_vx, sx, sy, sz, vx);
	interpolate(tex_vy, sx, sy, sz, vy);
	interpolate(tex_vz, sx, sy, sz, vz);

	hipMemcpy(sx, vx, sizeof(float)*len, hipMemcpyDeviceToDevice);
	hipMemcpy(sy, vy, sizeof(float)*len, hipMemcpyDeviceToDevice);
	hipMemcpy(sz, vz, sizeof(float)*len, hipMemcpyDeviceToDevice);

	coordinateImageKernel KERNEL_ARGS2(gridsize, blocksize) (d_sx, d_sy, d_sz, d3_dim, substractCoordinate);

	hipDestroyTextureObject(tex_vx);
	getCudaError("destroyTextureObject");
	hipDestroyTextureObject(tex_vy);
	getCudaError("destroyTextureObject");
	hipDestroyTextureObject(tex_vz);
	getCudaError("destroyTextureObject");
}

void LogDemonsRegGPU::iminterpolate(){
	dim3 blocksize;
	dim3 gridsize;

	blocksize = dim3(dim[0], 1, 1);
	gridsize = dim3(dim[1], dim[2], 1);
	coordinateImageKernel KERNEL_ARGS2(gridsize, blocksize) (d_sx, d_sy, d_sz, d3_dim, addCoordinate);

	interpolate(tex_mov, sx, sy, sz, deformedMoving);
}

void LogDemonsRegGPU::interpolate(hipTextureObject_t tex_I, float* d_sx, float* d_sy, float* d_sz, float* d_Ip){

	dim3 blocksize = dim3(8, 8, 8);
	dim3 gridsize = dim3(ceil(double(dim[0]) / double(blocksize.x)), ceil(double(dim[1]) / double(blocksize.y)), ceil(double(dim[2]) / double(blocksize.z)));

	if (debug){
		cout << "interpolating image..." << endl;
		cout << "blocksize: " << blocksize.x << " " << blocksize.y << " " << blocksize.z << " " << endl;
		cout << "gridsize: " << gridsize.x << " " << gridsize.y << " " << gridsize.z << " " << endl;
	}
	interpolateImageKernel KERNEL_ARGS2(gridsize, blocksize) (tex_I, d_sx, d_sy, d_sz, d_Ip, d3_dim);
	
	getCudaError("interpolateImageKernel");
	hipDeviceSynchronize();
	getCudaError("DeviceSynchorize interpolateImageKernel");
}



void LogDemonsRegGPU::expfield(){
	dim3 blocksize = dim3(1024, 1, 1);
	dim3 gridsize = dim3(ceil(float(len) / float(blocksize.x)), 1, 1);

	if (debug){
		cout << "Computing update kernel..." << endl;
		cout << "blocksize: " << blocksize.x << " " << blocksize.y << " " << blocksize.z << " " << endl;
		cout << "gridsize: " << gridsize.x << " " << gridsize.y << " " << gridsize.z << " " << endl;
	}
	normalizeVectorKernel KERNEL_ARGS2(gridsize, blocksize) (d_vx, d_vy, d_vz, d_normg2, len);
	getCudaError("normalizeVectorKernel");
	if (debug) {
		hipDeviceSynchronize();
		getCudaError("DeviceSynchorize normalizeVectorKernel");
	}

	float v2max = thrustFindMaxElement(d_normg2);
	int N = 0; 
	while (v2max > 0.5f){
		N++;
		v2max *= 0.5;
	}
	float scale = pow((float)2, -N);

	if (debug){
		cout << "First-order integration..." << endl;
		cout << "blocksize: " << blocksize.x << " 1 1 " << " " << endl;
		cout << "gridsize: " << gridsize.x << " 1 1 " << " " << endl;
	}

	scaleKernel KERNEL_ARGS2(gridsize, blocksize) (d_vx, d_vy, d_vz, d_sx, d_sy, d_sz, scale, len);

	//Recursive scaling and squaring
	printf("self-composing for %i times...\n", N);
	for (int i = 0; i < N; ++i){
		self_compose();
	}
}


float LogDemonsRegGPU::energy(){

	float reg_weight = (opt.sigma_i*opt.sigma_i) / (opt.sigma_x*opt.sigma_x);
	//pixel-wise energy
	dim3 blocksize = dim3(8, 8, 8);
	dim3 gridsize = dim3(ceil(double(dim[0]) / double(blocksize.x)), ceil(double(dim[1]) / double(blocksize.y)), ceil(double(dim[2]) / double(blocksize.z)));

	if (debug){
		cout << "Computing energy..." << endl;
		cout << "blocksize: " << blocksize.x << " " << blocksize.y << " " << blocksize.z << " " << endl;
		cout << "gridsize: " << gridsize.x << " " << gridsize.y << " " << gridsize.z << " " << endl;
	}
	energyKernel KERNEL_ARGS2(gridsize, blocksize) (d_fixed, d_deformedMoving, d_sx, d_sy, d_sz, d_en, reg_weight, d3_dim);

	getCudaError("energyKernel");
	hipDeviceSynchronize();
	getCudaError("DeviceSynchorize energyKernel");

	// thrust kicks in
	thrust::device_vector<float> devVec(d_en, d_en + len);
	float sum = thrust::reduce(devVec.begin(), devVec.end());

	return sum;
}